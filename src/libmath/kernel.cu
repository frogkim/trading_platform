#include "hip/hip_runtime.h"
﻿#include "libmath_header.h"
extern struct _dev_data dev_data;

bool isError(void)
{
    return g_last_cuda_error != hipSuccess;
}

char* getLastErrsorMessage_pChar(void)
{
    return g_error_message;
}

bool SetDevice(int index = 0)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    g_last_cuda_error = hipSetDevice(index);
    if (g_last_cuda_error != hipSuccess) {
        strcpy(g_error_message, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return false;
    }
    return true;
}

__global__ void _addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void _addWithQueueKernel(int* out, const int* queue)
{
    int i = threadIdx.x;
    out[i] = queue[0] + queue[1];
    
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t _addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (g_last_cuda_error != hipSuccess) {
        strcpy(g_error_message, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    _addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}


hipError_t _addWithCudaQueue(void)
{
    hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipMalloc((void**)&g_dev_data_int32.out, sizeof(int) * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        strcpy(g_error_message, "Failed to allocate cuda memory for out_vector");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&g_dev_data_int32.queue, sizeof(int) * sizeof(uint32_t) * g_dev_data_int32.queue_size);
    if (cudaStatus != hipSuccess) {
        strcpy(g_error_message, "Failed to allocate cuda memory for input_vector");
        return cudaStatus;
    }


    hipFree(g_dev_data_int32.out);
    hipFree(g_dev_data_int32.queue);
    return cudaStatus;
}
