#include "libmath_header.h"
extern struct _dev_data dev_data;

bool DeviceReset(void)
{
    hipError_t ret = hipDeviceReset();
    return (ret == hipSuccess) ? true : false;
}

bool addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    hipError_t ret = _addWithCuda(c, a, b, size);

    if (ret != hipSuccess) {
        return false;
    }

    return true;
}

bool addWithCudaBegin(int* out, const int* input_a, const int* input_b, unsigned int size)
{
    if (g_dev_data_int32.isContinue) {
        return false;
    }

    SetDevice(0);
    g_dev_data_int32.isContinue = true;
    g_dev_data_int32.out = out;
    g_dev_data_int32.queue[0] = input_a;
    g_dev_data_int32.queue[1] = input_a;
    g_dev_data_int32.queue_size = 2;
    return true;
}

bool addWithCudaContinue(const int* input)
{
    if (!g_dev_data_int32.isContinue) {
        // not initialized
        return false;
    }
    if (g_dev_data_int32.queue_size >= g_max_queue_size.uint32) {
        // reached maximum queue size
        return false;
    }

    g_dev_data_int32.queue[g_dev_data_int32.queue_size] = input;
    g_dev_data_int32.queue_size++;
    return true;
}
bool addWithCudaTerminate(void)
{
    if (!g_dev_data_int32.isContinue) {
        // not initialized
        strcpy(g_error_message, "Not initialized.");
        return false;
    }
    hipError_t ret = _addWithCudaQueue();

    g_dev_data_int32.isContinue = false;
    g_dev_data_int32.size = 0;
    g_dev_data_int32.out = 0;
    g_dev_data_int32.queue_size = 0;
    memset(g_dev_data_int32.queue, 0, sizeof(int*)* g_dev_data_int32.queue_size);
    
    if (ret != hipSuccess) {
        strcpy(g_error_message, "Other errors.");
        return false;
    }
    return true;
}
